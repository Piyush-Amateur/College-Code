#include <iostream>
#include <hip/hip_runtime.h>

// Kernel function for matrix multiplication
__global__ void matrixMultiply(int *A, int *B, int *C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < N && col < N) {
        int value = 0;
        for (int k = 0; k < N; ++k) {
            value += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = value;
    }
}

int main() {
    const int N = 1024; // Size of the matrix (N x N)
    size_t bytes = N * N * sizeof(int);
    
    int *A, *B, *C;
    int *d_A, *d_B, *d_C;

    // Allocate host memory
    A = (int*)malloc(bytes);
    B = (int*)malloc(bytes);
    C = (int*)malloc(bytes);

    // Initialize matrices
    for (int i = 0; i < N * N; ++i) {
        A[i] = i % 100;
        B[i] = (i * 2) % 100;
    }

    // Allocate device memory
    hipMalloc(&d_A, bytes);
    hipMalloc(&d_B, bytes);
    hipMalloc(&d_C, bytes);

    // Copy data from host to device
    hipMemcpy(d_A, A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, bytes, hipMemcpyHostToDevice);

    // Set up execution configuration
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((N + 15) / 16, (N + 15) / 16);

    // Launch kernel to multiply matrices
    matrixMultiply<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Copy result from device to host
    hipMemcpy(C, d_C, bytes, hipMemcpyDeviceToHost);

    // Check part of the result
    for (int i = 0; i < 10; ++i) {
        for (int j = 0; j < 10; ++j) {
            std::cout << "C[" << i << "][" << j << "] = " << C[i * N + j] << std::endl;
        }
    }

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    free(A);
    free(B);
    free(C);

    return 0;
}
