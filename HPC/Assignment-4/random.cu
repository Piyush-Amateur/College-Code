#include <iostream>
#include <hip/hip_runtime.h>


using namespace std;

#define BLOCK_SIZE 2

__global__ void gpuMM(float *A, float *B, float *C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        float sum = 0.f;
        for (int n = 0; n < N; ++n)
            sum += A[row * N + n] * B[n * N + col];
        C[row * N + col] = sum;
    }
}

int main() {
    int N = BLOCK_SIZE * 1;  // You can change multiplier from 1 to 2, 3, etc. for bigger matrices

    cout << "\nExecuting Matrix Multiplication\n";
    cout << "Matrix size: " << N << " x " << N << endl;

    // Host memory allocation
    float *hA = new float[N * N];
    float *hB = new float[N * N];
    float *hC = new float[N * N];

    // Initialize host matrices
    for (int i = 0; i < N * N; ++i) {
        hA[i] = 2.0f;
        hB[i] = 4.0f;
    }

    // Device memory allocation
    float *dA, *dB, *dC;
    size_t size = N * N * sizeof(float);
    hipMalloc(&dA, size);
    hipMalloc(&dB, size);
    hipMalloc(&dC, size);

    // Copy data to device
    hipMemcpy(dA, hA, size, hipMemcpyHostToDevice);
    hipMemcpy(dB, hB, size, hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocks((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (N + BLOCK_SIZE - 1) / BLOCK_SIZE);

    // Launch kernel
    gpuMM<<<blocks, threads>>>(dA, dB, dC, N);

    // Synchronize to wait for GPU to finish
    hipDeviceSynchronize();

    // Copy result back to host
    hipMemcpy(hC, dC, size, hipMemcpyDeviceToHost);

    // Print input matrices and result
    cout << "\nInput Matrix A:\n";
    for (int row = 0; row < N; ++row) {
        for (int col = 0; col < N; ++col)
            cout << hA[row * N + col] << " ";
        cout << endl;
    }

    cout << "\nInput Matrix B:\n";
    for (int row = 0; row < N; ++row) {
        for (int col = 0; col < N; ++col)
            cout << hB[row * N + col] << " ";
        cout << endl;
    }

    cout << "\nResultant Matrix C = A * B:\n";
    for (int row = 0; row < N; ++row) {
        for (int col = 0; col < N; ++col)
            cout << hC[row * N + col] << " ";
        cout << endl;
    }

    // Free memory
    delete[] hA;
    delete[] hB;
    delete[] hC;
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    cout << "\nFinished." << endl;
    return 0;
}
