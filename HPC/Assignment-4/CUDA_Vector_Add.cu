#include <iostream>
#include <hip/hip_runtime.h>

// Kernel function to add two vectors
__global__ void vectorAdd(int *A, int *B, int *C, int size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        C[index] = A[index] + B[index];
    }
}

int main() {
    const int SIZE = 1000000; // Size of the vectors
    int *A, *B, *C;
    int *d_A, *d_B, *d_C;
    size_t bytes = SIZE * sizeof(int);

    // Allocate host memory
    A = (int*)malloc(bytes);
    B = (int*)malloc(bytes);
    C = (int*)malloc(bytes);

    // Initialize vectors
    for (int i = 0; i < SIZE; ++i) {
        A[i] = i;
        B[i] = i * 2;
    }

    // Allocate device memory
    hipMalloc(&d_A, bytes);
    hipMalloc(&d_B, bytes);
    hipMalloc(&d_C, bytes);

    // Copy data from host to device
    hipMemcpy(d_A, A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, bytes, hipMemcpyHostToDevice);

    // Launch kernel to add vectors
    int blockSize = 256;
    int gridSize = (SIZE + blockSize - 1) / blockSize;
    vectorAdd<<<gridSize, blockSize>>>(d_A, d_B, d_C, SIZE);

    // Copy result from device to host
    hipMemcpy(C, d_C, bytes, hipMemcpyDeviceToHost);

    // Check the result
    for (int i = 0; i < 10; ++i) {
        std::cout << "C[" << i << "] = " << C[i] << std::endl;
    }

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    free(A);
    free(B);
    free(C);

    return 0;
}
